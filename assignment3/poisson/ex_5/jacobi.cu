#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "alloc3d.h"

__global__ void jacobi(int N, double ***u_new,double ***u_old, double***f) {
    double delta;
    double delta_sq;
    delta = 2.0/(N + 1.0);
    delta_sq = delta * delta;
    int z, y, x;
    double div = 1.0/6.0;
    //perform Jacobi iterations
    for(z=1;z<N+1;z++){
        for(y=1;y<N+1;y++){
            for(x=1;x<N+1;x++){
                u_new[z][y][x] = div * (u_old[z-1][y][x] + \
                                        u_old[z+1][y][x] + \
                                        u_old[z][y-1][x] + \
                                        u_old[z][y+1][x] + \
                                        u_old[z][y][x-1] + \
                                        u_old[z][y][x+1] + \
                                        delta_sq * f[z][y][x]);
            }
        }
    }
}
