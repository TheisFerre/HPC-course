
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define min(x,y) (((x) < (y)) ? (x) : (y))

extern "C" {
    #include <cblas.h>
    void matmult_nat(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (mkn)
        for(m = 0; m < M; m++){
            for(k = 0; k < K; k++){
                for(n = 0; n < N; n++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }
    }

    void matmult_lib(int M, int N, int K, double *A, double *B, double *C){

        // first dimension of A (m X K)
        int lda = K;
        // first dimension of B (K X N)
        int ldb = N;
        // first dimension of C (M X N)
        int ldc = N;

        double time, elapsed;

        time = omp_get_wtime();
        // call cblas library
        cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, 1.0, A, lda, B, ldb, 0.0, C, ldc);
        elapsed = omp_get_wtime() - time;
    }

    void matmult_mkn(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (mkn)
        for(m = 0; m < M; m++){
            for(k = 0; k < K; k++){
                for(n = 0; n < N; n++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }
    }

    void matmult_mnk(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (mnk)
        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                for(k = 0; k < K; k++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }

    }

    void matmult_kmn(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (kmn)
        for(k = 0; k < K; k++){
            for(m = 0; m < M; m++){
                for(n = 0; n < N; n++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }

    }

    void matmult_knm(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (knm)
        for(k = 0; k < K; k++){
            for(n = 0; n < N; n++){
                for(m = 0; m < M; m++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }
    }

    void matmult_nkm(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (nkm)
        for(n = 0; n < N; n++){
            for(k = 0; k < K; k++){
                for(m = 0; m < M; m++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }

    }

    void matmult_nmk(int M, int N, int K, double *A, double *B, double *C) {

        int m, n, k;

        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }
        
        //Triple for loop for calculation (nmk)
        for(n = 0; n < N; n++){
            for(m = 0; m < M; m++){
                for(k = 0; k < K; k++){
                    C[m * N + n] += A[m * K + k] * B[k * N + n];
                }
            }
        }
    }

    void matmult_blk(int M, int N, int K, double *A, double *B, double *C, int bs){
        int kk,nn,n,k,m;
        
        for(m = 0; m < M; m++){
            for(n = 0; n < N; n++){
                C[m * N + n] = 0;
            }
        }

        int m0, k0, n0;

        for (m0 = 0; m0 < M; m0 += bs)
        { int minm0=min(m0 + bs,M);
            for (k0 = 0; k0 < K; k0 += bs)
            { int mink0=min(k0 + bs,K);
                for (n0 = 0; n0 < N; n0 += bs)
                {  int minn0=min(n0 + bs,N);
                    for (m = m0; m < minm0; m++)
                    {
                        for (k = k0; k < mink0; k++)
                        {
                            for (n = n0; n <minn0; n++)
                            {
                                C[m * N + n] += A[m * K + k] * B[k * N + n];
                            }
                        }
                    }
                }
            }
        }
    }

}