#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 16
#define IDX2C(i,j,ld) (((j)*(ld))+(i) // https://docs.nvidia.com/cuda/cublas/index.html compatible way to index 2d matrices in C for cublas

extern "C" {
   void matmult_gpulib(int M, int N, int K, double *A_h, double *B_h, double *C_h); 
}

void matmult_gpulib(int M, int N, int K, double *A_h, double *B_h, double *C_h)
{
    double *A_d;
    double *B_d;
    double *C_d;

    int A_size = M * K * sizeof(double);
    int B_size = K * N * sizeof(double);
    int C_size = M * N * sizeof(double);

    hipMalloc((void **)&A_d, A_size);
    hipMalloc((void **)&B_d, B_size);
    hipMalloc((void **)&C_d, C_size);

    // transfer data to cuda
    hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

    // initiate threads (how do we size them?)
    // Initialize number of blocks and threads
    // int BLOCK_SIZE = 16;
    dim3 numOfThreadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    int xSize = ceil((double)(N + numOfThreadsPerBlock.x - 1) / (double)numOfThreadsPerBlock.x);
    int ySize = ceil((double)(M + numOfThreadsPerBlock.y - 1) / (double)numOfThreadsPerBlock.y);
    dim3 numOfBlocks(xSize, ySize);

    /* CALL TO: library routine */
    // A = M X K
    // B = K X N
    // C = M X N

    // Column Major: Leading Dimension (ld) = # of rows 
    // first dimension of A (M X K)
    int lda = K;
    // first dimension of B (K X N)
    int ldb = N;
    // first dimension of C (M X N)
    int ldc = N;

    const double alf = 1.0;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // call cblas library
    // cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, 1.0, A, lda, B, ldb, 0.0, C, ldc);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, alpha, A_d, lda, B_d, ldb, beta, C_d, ldc);

    // destroy handle
    hipblasDestroy(handle);
    /*END OF CALL*/

    // checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}