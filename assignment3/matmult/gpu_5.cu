#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

// Thread block size
#define BLOCK_SIZE 4

// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride; 
    double* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

__global__ void gpu5_kernel(Matrix A, Matrix B, Matrix C){
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e) {
            Cvalue += As[row][e] * Bs[e][col];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // // Write Csub to device memory
    // // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
    
}

extern "C" {
    void matmult_gpu5(int M, int N, int K, double *A_h, double *B_h, double *C_h){

        // initialize matrix structs
        Matrix A, B, C;
        A.width = K; A.height = M; A.elements = A_h;
        B.width = N; B.height = K; B.elements = B_h;
        C.width = N; C.height = M; C.elements = C_h;

        // Load A and B to device memory
        Matrix d_A;
        d_A.width = d_A.stride = A.width; d_A.height = A.height;
        size_t size = A.width * A.height * sizeof(double);
        hipMalloc(&d_A.elements, size);
        hipMemcpy(d_A.elements, A.elements, size,
                hipMemcpyHostToDevice);

        Matrix d_B;
        d_B.width = d_B.stride = B.width; d_B.height = B.height;
        size = B.width * B.height * sizeof(double);
        hipMalloc(&d_B.elements, size);
        hipMemcpy(d_B.elements, B.elements, size,
                hipMemcpyHostToDevice);

        // Allocate C in device memory
        Matrix d_C;
        d_C.width = d_C.stride = C.width; d_C.height = C.height;
        size = C.width * C.height * sizeof(double);
        hipMalloc(&d_C.elements, size);

        // Invoke kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
        gpu5_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

        checkCudaErrors(hipDeviceSynchronize());

        // Read C from device memory
        hipMemcpy(C.elements, d_C.elements, size,
                hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
    }
}