#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
__global__ void gpu3_kernel(int M, int N, int K, double *A_d, double *B_d, double *C_d)
{

    int ROW1 = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int ROW2 = ROW1 + 1;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    // A = M X K
    // B = K X N
    // C = M X N

    if (ROW2 < M && COL < N)
    {
        int i, j;
        double sum_val1 = 0;
        double sum_val2 = 0;
        for (i = 0; i < K; i++)
        {
            sum_val1 += A_d[ROW1 * K + i] * B_d[i * N + COL];
            sum_val2 += A_d[ROW2 * K + i] * B_d[i * N + COL];
        }
        C_d[ROW1 * N + COL] = sum_val1;
        C_d[ROW2 * N + COL] = sum_val2;
    }

    else if (ROW1 < M && COL < N)
    {
        int i, j;
        double sum_val1 = 0;
        for (i = 0; i < K; i++)
        {
            sum_val1 += A_d[ROW1 * K + i] * B_d[i * N + COL];
        }
        C_d[ROW1 * N + COL] = sum_val1;
    }
}
extern "C"
{
    void matmult_gpu3(int M, int N, int K, double *A_h, double *B_h, double *C_h)
    {
        double *A_d;
        double *B_d;
        double *C_d;

        double time, elapsed;
        double transfer_time, transfer_elabsed;

        int A_size = M * K * sizeof(double);
        int B_size = K * N * sizeof(double);
        int C_size = M * N * sizeof(double);

        hipMalloc((void **)&A_d, A_size);
        hipMalloc((void **)&B_d, B_size);
        hipMalloc((void **)&C_d, C_size);

        // transfer data to cuda
        transfer_time = omp_get_wtime();
        hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
        hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);
        transfer_elabsed = omp_get_wtime() - transfer_time;

        // initiate threads (how do we size them?)
        // Initialize number of blocks and threads
        // M / BLOCK_SIZE has to be greater than or equal to 1
        dim3 THREADS_BLOCK(BLOCK_SIZE, BLOCK_SIZE);
        int xSize = (N + BLOCK_SIZE - 1) / THREADS_BLOCK.x;
        int ySize = (M + BLOCK_SIZE - 1) / THREADS_BLOCK.y;
        dim3 GRIDSIZE(xSize, ySize);

        time = omp_get_wtime();
 
        gpu3_kernel<<<GRIDSIZE, THREADS_BLOCK>>>(M, N, K, A_d, B_d, C_d);
        checkCudaErrors(hipDeviceSynchronize()); 
        
        elapsed = omp_get_wtime() - time;

        //printf("Kernel_time\t");
        //printf("Transfer_time\n");
        printf("%f\t%f\n", elapsed, transfer_elabsed);

        hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
        hipFree(A_d);
        hipFree(B_d);
        hipFree(C_d);
    }
}