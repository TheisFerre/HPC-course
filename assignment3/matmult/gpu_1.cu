#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
__global__ void gpu1_kernel(int M, int N, int K, double *A_d, double *B_d, double *C_d)
{

    // single thread to compute all data
    int m, n, k;
    
    for (n = 0; n < N; n++)
    {
        for (m = 0; m < M; m++)
        {
            double sum_val = 0;
            for (k = 0; k < K; k++)
            {
                 sum_val += A_d[m * K + k] * B_d[k * N + n];
            }
            C_d[m * N + n] = sum_val;
        }   
    }
}
extern "C"
{
    void matmult_gpu1(int M, int N, int K, double *A_h, double *B_h, double *C_h)
    {
        double *A_d;
        double *B_d;
        double *C_d;

        double time, elapsed;
        double transfer_time, transfer_elabsed;

        int A_size = M * K * sizeof(double);
        int B_size = K * N * sizeof(double);
        int C_size = M * N * sizeof(double);

        hipMalloc((void **)&A_d, A_size);
        hipMalloc((void **)&B_d, B_size);
        hipMalloc((void **)&C_d, C_size);

        // transfer data to cuda
        transfer_time = omp_get_wtime();
        hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
        hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);
        hipMemcpy(C_d, C_h, C_size, hipMemcpyHostToDevice);
        transfer_elabsed = omp_get_wtime() - transfer_time;

        // Initialize number of blocks and threads
        dim3 THREADS_BLOCK(1, 1);
        dim3 GRIDSIZE(1, 1);

        time = omp_get_wtime();
 
        gpu1_kernel<<<GRIDSIZE, THREADS_BLOCK>>>(M, N, K, A_d, B_d, C_d);
        checkCudaErrors(hipDeviceSynchronize());
        
        elapsed = omp_get_wtime() - time;

        transfer_time = omp_get_wtime();
        hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
        transfer_elabsed += omp_get_wtime() - transfer_time;

        //printf("Kernel_time\t");
        //printf("Transfer_time\n");
        printf("%f\t%f\n", elapsed, transfer_elabsed);

        hipFree(A_d);
        hipFree(B_d);
        hipFree(C_d);
    }
}